#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

// Estructura para almacenar elementos no nulos de la matriz dispersa
typedef struct {
    int fila;
    int columna;
    double valor;
} Elemento;

// Estructura para la matriz dispersa
typedef struct {
    Elemento* elementos;
    int num_elementos;
    int filas;
    int columnas;
} MatrizDispersa;

// Kernel CUDA optimizado para multiplicar matriz dispersa por vector
__global__ void matriz_vector_kernel(Elemento* elementos, int num_elementos, 
                                   double* vector, double* resultado, int filas) {
    // Calcular índice global del hilo
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Memoria compartida para el vector
    extern __shared__ double s_vector[];
    
    // Cargar parte del vector en memoria compartida
    int tid = threadIdx.x;
    int stride = blockDim.x;
    for (int i = tid; i < filas; i += stride) {
        s_vector[i] = vector[i];
    }
    __syncthreads();
    
    // Cada hilo procesa un elemento no nulo de la matriz
    if (idx < num_elementos) {
        int fila = elementos[idx].fila;
        int columna = elementos[idx].columna;
        double valor = elementos[idx].valor;
        
        // Multiplicar el elemento por el correspondiente del vector
        // y usamos atomicAdd para evitar condiciones de carrera
        atomicAdd(&resultado[fila], valor * s_vector[columna]);
    }
}

// Función optimizada para multiplicar matriz dispersa por vector usando CUDA
void multiplicar_matriz_vector_cuda(MatrizDispersa* matriz, double* vector, double* resultado) {
    // Variables para dispositivo (GPU)
    Elemento* d_elementos;
    double* d_vector;
    double* d_resultado;
    
    // Reservar memoria en GPU
    hipMalloc((void**)&d_elementos, matriz->num_elementos * sizeof(Elemento));
    hipMalloc((void**)&d_vector, matriz->columnas * sizeof(double));
    hipMalloc((void**)&d_resultado, matriz->filas * sizeof(double));
    
    // Inicializar el vector resultado a ceros en GPU
    hipMemset(d_resultado, 0, matriz->filas * sizeof(double));
    
    // Crear streams para transferencias asíncronas
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    
    // Copiar datos de CPU a GPU de forma asíncrona
    hipMemcpyAsync(d_elementos, matriz->elementos, 
                    matriz->num_elementos * sizeof(Elemento), 
                    hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_vector, vector, 
                    matriz->columnas * sizeof(double), 
                    hipMemcpyHostToDevice, stream2);
    
    // Esperar a que terminen las transferencias
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    
    // Configurar la ejecución del kernel
    int blockSize = 256;
    int numBlocks = (matriz->num_elementos + blockSize - 1) / blockSize;
    int sharedMemSize = matriz->columnas * sizeof(double);
    
    // Ejecutar el kernel
    matriz_vector_kernel<<<numBlocks, blockSize, sharedMemSize>>>
        (d_elementos, matriz->num_elementos, d_vector, d_resultado, matriz->filas);
    
    // Esperar a que termine el kernel
    hipDeviceSynchronize();
    
    // Copiar el resultado de GPU a CPU
    hipMemcpy(resultado, d_resultado, matriz->filas * sizeof(double), 
               hipMemcpyDeviceToHost);
    
    // Liberar memoria en GPU
    hipFree(d_elementos);
    hipFree(d_vector);
    hipFree(d_resultado);
    
    // Destruir streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
}

// Función para leer la matriz dispersa
MatrizDispersa* leer_matriz_dispersa(FILE *archivo, int *filas, int *columnas) {
    if (fscanf(archivo, "%d %d", filas, columnas) != 2) {
        printf("Error al leer dimensiones\n");
        return NULL;
    }

    // Reservamos memoria para la matriz
    MatrizDispersa* matriz = (MatrizDispersa*)malloc(sizeof(MatrizDispersa));
    if (!matriz) {
        printf("Error al reservar memoria para la matriz\n");
        return NULL;
    }

    // Estimamos el número de elementos no nulos (10% de la matriz)
    int num_elementos = (*filas * *columnas) / 10;
    matriz->elementos = (Elemento*)malloc(num_elementos * sizeof(Elemento));
    if (!matriz->elementos) {
        printf("Error al reservar memoria para los elementos\n");
        free(matriz);
        return NULL;
    }

    // Leemos y guardamos solo elementos no nulos
    int idx = 0;
    double valor;
    for (int i = 0; i < *filas; i++) {
        for (int j = 0; j < *columnas; j++) {
            if (fscanf(archivo, "%lf", &valor) != 1) {
                printf("Error al leer elemento [%d,%d]\n", i, j);
                free(matriz->elementos);
                free(matriz);
                return NULL;
            }
            if (valor != 0) {
                if (idx >= num_elementos) {
                    // Redimensionar si es necesario
                    num_elementos *= 2;
                    Elemento* temp = realloc(matriz->elementos, num_elementos * sizeof(Elemento));
                    if (!temp) {
                        printf("Error al redimensionar memoria\n");
                        free(matriz->elementos);
                        free(matriz);
                        return NULL;
                    }
                    matriz->elementos = temp;
                }
                matriz->elementos[idx].fila = i;
                matriz->elementos[idx].columna = j;
                matriz->elementos[idx].valor = valor;
                idx++;
            }
        }
    }

    // Ajustar al tamaño real
    if (idx < num_elementos) {
        Elemento* temp = realloc(matriz->elementos, idx * sizeof(Elemento));
        if (temp) {
            matriz->elementos = temp;
        }
    }

    matriz->num_elementos = idx;
    matriz->filas = *filas;
    matriz->columnas = *columnas;
    return matriz;
}

// Función para leer el vector
double* leer_vector(FILE *archivo, int *dimension) {
    if (fscanf(archivo, "%d", dimension) != 1) {
        printf("Error al leer dimensión del vector\n");
        return NULL;
    }

    double *vector = (double *)malloc(*dimension * sizeof(double));
    if (!vector) {
        printf("Error al reservar memoria para el vector\n");
        return NULL;
    }

    // Leer todo el vector de una vez
    for (int i = 0; i < *dimension; i++) {
        if (fscanf(archivo, "%lf", &vector[i]) != 1) {
            printf("Error al leer elemento %d del vector\n", i);
            free(vector);
            return NULL;
        }
    }

    return vector;
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Uso: %s <archivo_matriz> <archivo_vector>\n", argv[0]);
        return 1;
    }
    
    // Declarar variables
    MatrizDispersa* matriz = NULL;
    double* vector = NULL;
    double* resultado = NULL;
    int filas = 0, columnas = 0, dim_vector = 0;
    
    // Abrir archivos
    FILE *archivo_matriz = fopen(argv[1], "r");
    FILE *archivo_vector = fopen(argv[2], "r");
    
    if (!archivo_matriz || !archivo_vector) {
        printf("Error al abrir los archivos\n");
        if (archivo_matriz) fclose(archivo_matriz);
        if (archivo_vector) fclose(archivo_vector);
        return 1;
    }
    
    // Leer matriz dispersa y vector
    matriz = leer_matriz_dispersa(archivo_matriz, &filas, &columnas);
    vector = leer_vector(archivo_vector, &dim_vector);
    
    fclose(archivo_matriz);
    fclose(archivo_vector);
    
    if (!matriz || !vector) {
        if (matriz) {
            free(matriz->elementos);
            free(matriz);
        }
        if (vector) free(vector);
        return 1;
    }
    
    // Verificar dimensiones
    if (columnas != dim_vector) {
        printf("Error: Las dimensiones no son compatibles\n");
        free(matriz->elementos);
        free(matriz);
        free(vector);
        return 1;
    }
    
    // Reservar memoria para el resultado
    resultado = (double*)malloc(filas * sizeof(double));
    if (!resultado) {
        printf("Error al reservar memoria para el resultado\n");
        free(matriz->elementos);
        free(matriz);
        free(vector);
        return 1;
    }
    
    // Medir tiempo de ejecución
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    // Realizar multiplicación
    multiplicar_matriz_vector_cuda(matriz, vector, resultado);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tiempo;
    hipEventElapsedTime(&tiempo, start, stop);
    
    // Imprimir tiempo de ejecución
    printf("Tiempo de ejecución: %.6f segundos\n", tiempo / 1000.0);
    
    // Liberar memoria
    free(matriz->elementos);
    free(matriz);
    free(vector);
    free(resultado);
    
    // Destruir eventos CUDA
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}
